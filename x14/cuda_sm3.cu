#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include <cuda_helper.h>
#include <miner.h>


#define  F(x, y, z) (((x) ^ (y) ^ (z)))
#define FF(x, y, z) (((x) & (y)) | ((x) & (z)) | ((y) & (z)))
#define GG(x, y, z) ((z)  ^ ((x) & ((y) ^ (z))))

#define P0(x) x ^ ROTL32(x,  9) ^ ROTL32(x, 17)
#define P1(x) x ^ ROTL32(x, 15) ^ ROTL32(x, 23)

__device__
void sm3_compress2(uint32_t digest[8], unsigned char block[64]){
	uint32_t tt1, tt2, i, t, ss1, ss2, x, y;
	uint32_t w[68];
	uint32_t a = digest[0];
	uint32_t b = digest[1];
	uint32_t c = digest[2];
	uint32_t d = digest[3];
	uint32_t e = digest[4];
	uint32_t f = digest[5];
	uint32_t g = digest[6];
	uint32_t h = digest[7];


	const uint32_t *pblock = (const uint32_t *)block;

	for (i = 0; i<16; i++) {
		w[i] = cuda_swab32(pblock[i]);
	}

	for (i = 16; i<68; i++) {
		x = ROTL32(w[i - 3], 15);
		y = ROTL32(w[i - 13], 7);

		x ^= w[i - 16];
		x ^= w[i - 9];
		y ^= w[i - 6];

		w[i] = P1(x) ^ y;
	}

	for (i = 0; i<64; i++) {

		t = (i < 16) ? 0x79cc4519 : 0x7a879d8a;

		ss2 = ROTL32(a, 12);
		ss1 = ROTL32(ss2 + e + ROTL32(t, i), 7);
		ss2 ^= ss1;

		tt1 = d + ss2 + (w[i] ^ w[i + 4]);
		tt2 = h + ss1 + w[i];

		if (i < 16) {
			tt1 += F(a, b, c);
			tt2 += F(e, f, g);
		}
		else {
			tt1 += FF(a, b, c);
			tt2 += GG(e, f, g);
		}
		d = c;
		c = ROTL32(b, 9);
		b = a;
		a = tt1;
		h = g;
		g = ROTL32(f, 19);
		f = e;
		e = P0(tt2);
	}


	digest[0] ^= a;
	digest[1] ^= b;
	digest[2] ^= c;
	digest[3] ^= d;
	digest[4] ^= e;
	digest[5] ^= f;
	digest[6] ^= g;
	digest[7] ^= h;

}


/***************************************************/
// GPU Hash Function
__global__ void x14_sm3_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	__syncthreads();

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);
		int hashPosition = nounce - startNounce;

		uint32_t digest[8];
		digest[0] = 0x7380166F;
		digest[1] = 0x4914B2B9;
		digest[2] = 0x172442D7;
		digest[3] = 0xDA8A0600;
		digest[4] = 0xA96F30BC;
		digest[5] = 0x163138AA;
		digest[6] = 0xE38DEE4D;
		digest[7] = 0xB0FB0E4E;

		unsigned char *pHash = (unsigned char *)&g_hash[hashPosition << 3];
		sm3_compress2(digest, pHash);

		unsigned char block[64] = {0};

		block[0] = 0x80;
		uint32_t *count = (uint32_t *)(block + 64 - 8);

		count[0] = cuda_swab32(1 >> 23);
		count[1] = cuda_swab32((1 << 9) + (0 << 3));

		sm3_compress2(digest, block);

		uint32_t *outpHash = (uint32_t*)&g_hash[hashPosition << 3]; // [8 * hashPosition];

		for (int i = 0; i < 8; i++)
			outpHash[i] = cuda_swab32(digest[i]);

		for (int i = 8; i < 16; i++)
			outpHash[i] = 0;
	}
}

__host__ void x14_sm3_cpu_init(int thr_id, uint32_t threads)
{
}

// #include <stdio.h>
__host__ void x14_sm3_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;

	x14_sm3_gpu_hash_64 << <grid, block, shared_size >> >(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
	MyStreamSynchronize(NULL, order, thr_id);
}